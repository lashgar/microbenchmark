#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <openacc.h>
#define IPMACC_MAX1(A)   (A)
#define IPMACC_MAX2(A,B) (A>B?A:B)
#define IPMACC_MAX3(A,B,C) (A>B?(A>C?A:(B>C?B:C)):(B>C?C:B))
#include <hip/hip_runtime.h>

__global__ void __generated_kernel_region_0(int chunk,int* dummy2,int* dummy1,int nthread,long long unsigned int* time);

int main()
{
    int i, j, k; int nthread = NTHD, iteration = 6, chunk = 128 / sizeof(int); 
//    int dummy1 [(nthread) * chunk * iteration * 2], dummy2 [(nthread) * chunk * iteration * 2]; long long unsigned int time [nthread];
    int *dummy1=(int*)malloc((nthread) * chunk * iteration * 2*sizeof(int));
    int *dummy2=(int*)malloc((nthread) * chunk * iteration * 2*sizeof(int));
    long long unsigned int *time=(long long unsigned int*)malloc(nthread*sizeof(long long unsigned int));


    ipmacc_prompt((char*)"IPMACC: memory allocation dummy2\n");
    acc_create((void*)dummy2,( nthread ) *  chunk  *  iteration  *  2*sizeof(int));
    ipmacc_prompt((char*)"IPMACC: memory allocation dummy1\n");
    acc_create((void*)dummy1,( nthread ) *  chunk  *  iteration  *  2*sizeof(int));
    ipmacc_prompt((char*)"IPMACC: memory allocation time\n");
    acc_create((void*)time,nthread*sizeof(long long unsigned int));
    ipmacc_prompt((char*)"IPMACC: memory copyin dummy2\n");
    acc_copyin((void*)dummy2,( nthread ) *  chunk  *  iteration  *  2*sizeof(int));
    ipmacc_prompt((char*)"IPMACC: memory copyin dummy1\n");
    acc_copyin((void*)dummy1,( nthread ) *  chunk  *  iteration  *  2*sizeof(int));
    ipmacc_prompt((char*)"IPMACC: memory copyin time\n");
    acc_copyin((void*)time,nthread*sizeof(long long unsigned int));


    {





        /* kernel call statement [0, -1]*/
        if (getenv("IPMACC_VERBOSE")) printf("IPMACC: Launching kernel 0 > gridDim: %d\tblockDim: %d\n",(((abs((int)((nthread))-0))/(1)))/512+1,512);
        //__generated_kernel_region_0<<<(((abs((int)((nthread))-0))/(1)))/512+1,512>>>(
        __generated_kernel_region_0<<<1,nthread>>>(
                chunk,
                (int*)acc_deviceptr((void*)dummy2),
                (int*)acc_deviceptr((void*)dummy1),
                nthread,
                (long long unsigned int*)acc_deviceptr((void*)time));
        /* kernel call statement*/
        if (getenv("IPMACC_VERBOSE")) printf("IPMACC: Synchronizing the region with host\n");
        hipDeviceSynchronize();



    }
    ipmacc_prompt((char*)"IPMACC: memory copyout dummy2\n");
    acc_copyout_and_keep((void*)dummy2,( nthread ) *  chunk  *  iteration  *  2*sizeof(int));
    ipmacc_prompt((char*)"IPMACC: memory copyout dummy1\n");
    acc_copyout_and_keep((void*)dummy1,( nthread ) *  chunk  *  iteration  *  2*sizeof(int));
    ipmacc_prompt((char*)"IPMACC: memory copyout time\n");
    acc_copyout_and_keep((void*)time,nthread*sizeof(long long unsigned int));



    for (i = 0; i < nthread; i += 32) {
        printf("%llu	", time [i]);
    }
    for (; i < 512; i += 32) {
        printf("%llu	", 0);
    }
    return 0;
}


__device__ int __smc_select_0_dummy1(int index, int down, int up, int* g_array, int* s_array, int vector_size, int pivot, int before){
    // the pragmas are well-set. do not check the boundaries.
    return s_array[index-(vector_size*blockIdx.x)+before-pivot];
}

__device__ void __smc_write_0_dummy1(int index, int down, int up, int* g_array, int* s_array, int vector_size, int pivot, int before,int value){
    // the pragmas are well-set. do not check the boundaries.
    s_array[index-(vector_size*blockIdx.x)+before-pivot]=value;
}
__global__ void __generated_kernel_region_0(int chunk,int* dummy2,int* dummy1,int nthread,long long unsigned int* time){
    int __kernel_getuid=threadIdx.x+blockIdx.x*blockDim.x;
    int i;

    /* declare the shared memory of dummy1 */
    //__shared__ int __kernel_smc_var_data_dummy1[512+1+1];
    //__shared__ unsigned char __kernel_smc_var_tag_dummy1[512+1+1];
    __shared__ int __kernel_smc_var_data_dummy1[1024+1+1];
    __shared__ unsigned char __kernel_smc_var_tag_dummy1[1024+1+1];
    {
        int iterator_of_smc=0;
        for(iterator_of_smc=threadIdx.x; iterator_of_smc<(512+1+1); iterator_of_smc+=blockDim.x){
            __kernel_smc_var_data_dummy1[iterator_of_smc]=0;
            __kernel_smc_var_tag_dummy1[iterator_of_smc]=0;
        }
        __syncthreads();
    }
    {
        {
            {
                i=0+(__kernel_getuid);
                if( i < nthread)
                { // opened for smc fetch
                    { // fetch begins
                        int kk;
                        __syncthreads();
                        for(int kk=threadIdx.x; kk<(512+1+1); kk+=blockDim.x)
                        {
                            int idx=blockIdx.x*512+kk-1+0;
                            if(idx<(nthread) && idx>=(0))
                            {
                                __kernel_smc_var_data_dummy1[kk]=dummy1[idx];
                                __kernel_smc_var_tag_dummy1[kk]=1;
                            }
                        }
                        __syncthreads();
                    } // end of fetch
#define dummy1(index) __smc_select_0_dummy1(index, (blockIdx.x*512)-(1), ((blockIdx.x+1)*512)+(1), dummy1, __kernel_smc_var_data_dummy1, 512, 0, 1)


                    {
                        if (i < nthread) {
                            long long unsigned int tick, tock; unsigned long long int d1 = threadIdx.x / 32; unsigned long long int d2 = threadIdx.x / (DISP * 32); unsigned long long int addr1 = (unsigned long long int)dummy2 + ((threadIdx.x) + 32 * (d2 - d1) + ((blockDim.x / DISP) * 1)) * chunk * sizeof(int);
                            unsigned long long int addr2 = (unsigned long long int)dummy2 + ((threadIdx.x) + 32 * (d2 - d1) + ((blockDim.x / DISP) * 2)) * chunk * sizeof(int);
                            unsigned long long int addr3 = (unsigned long long int)dummy2 + ((threadIdx.x) + 32 * (d2 - d1) + ((blockDim.x / DISP) * 3)) * chunk * sizeof(int);
                            unsigned long long int addr4 = (unsigned long long int)dummy2 + ((threadIdx.x) + 32 * (d2 - d1) + ((blockDim.x / DISP) * 4)) * chunk * sizeof(int);
                            unsigned long long int addr5 = (unsigned long long int)dummy2 + ((threadIdx.x) + 32 * (d2 - d1) + ((blockDim.x / DISP) * 5)) * chunk * sizeof(int);
                            unsigned long long int addr6 = (unsigned long long int)dummy2 + ((threadIdx.x) + 32 * (d2 - d1) + ((blockDim.x / DISP) * 6)) * chunk * sizeof(int);
                            __syncthreads();
                            __smc_write_0_dummy1(i, (blockIdx.x*512)-(1), ((blockIdx.x+1)*512)+(1), dummy1, __kernel_smc_var_data_dummy1, 512, 0, 1, addr1 + addr2 + addr3 + addr4 + addr5 + addr6 + 0);
                            __syncthreads();
                            __syncthreads();
                            tick = clock();
                            int reg1; asm ("ld.global.s32 %0, [%1+0];" : "=r" (reg1) : "l" (addr1));
                            int reg2; asm ("ld.global.s32 %0, [%1+0];" : "=r" (reg2) : "l" (addr2));
                            int reg3; asm ("ld.global.s32 %0, [%1+0];" : "=r" (reg3) : "l" (addr3));
                            int reg4; asm ("ld.global.s32 %0, [%1+0];" : "=r" (reg4) : "l" (addr4));
                            int reg5; asm ("ld.global.s32 %0, [%1+0];" : "=r" (reg5) : "l" (addr5));
                            int reg6; asm ("ld.global.s32 %0, [%1+0];" : "=r" (reg6) : "l" (addr6));
                            reg1 = reg1 + reg2 + reg3 + reg4 + reg5 + reg6 + 0;
                            __syncthreads();
                            tock = clock();
                            __syncthreads();
                            __smc_write_0_dummy1(0, (blockIdx.x*512)-(1), ((blockIdx.x+1)*512)+(1), dummy1, __kernel_smc_var_data_dummy1, 512, 0, 1, reg1 + dummy1 (0));
                            __syncthreads();
                            time [i] = (tock - tick);
                        }
                    }
                    { // writeback begins
                        int kk;
                        __syncthreads();
                        for(int kk=threadIdx.x; kk<(512+1+1); kk+=blockDim.x)
                        {
                            int idx=blockIdx.x*512+kk-1+0;
                            if(idx<(nthread) && idx>=(0))
                            {
                                dummy1[idx]=__kernel_smc_var_data_dummy1[kk];
                            }
                        }
                        __syncthreads();
                    } // end of writeback

#undef dummy1

                } // closed for smc fetch end

            }
        }
    }
}

